#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         cuda_kmeans.cu  (CUDA version)                            */
/*   Description:  Implementation of simple k-means clustering algorithm     */
/*                 This program takes an array of N data objects, each with  */
/*                 M coordinates and performs a k-means clustering given a   */
/*                 user-provided value of the number of clusters (K). The    */
/*                 clustering results are saved in 2 arrays:                 */
/*                 1. a returned array of size [K][N] indicating the center  */
/*                    coordinates of K clusters                              */
/*                 2. membership[N] stores the cluster center ids, each      */
/*                    corresponding to the cluster a data object is assigned */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department, Northwestern University                        */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

    return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
__host__ __device__ inline static
float euclid_dist_2(int    numCoords,
                    int    numObjs,
                    float *objects,     // [numCoords][numObjs]
                    int    objectId,
                    float *center)      // [numCoords]
{
    int i;
    float ans=0.0;

    for (i = 0; i < numCoords; i++) {
        ans += (objects[numObjs * i + objectId] - center[i]) *
               (objects[numObjs * i + objectId] - center[i]);
    }

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,           //  [numObjs][numCoords]
                          float *deviceClusters,    //  [numClusters][numCoords]
                          int *membership,          //  [numObjs]
                          int *intermediates)
{
    extern __shared__ char sharedMemory[];

    //  The type chosen for membershipChanged must be large enough to support
    //  reductions! There are blockDim.x elements, one for each thread in the
    //  block.
    unsigned char *membershipChanged = (unsigned char *)sharedMemory;
    float *clusters = (float *)(sharedMemory + blockDim.x);

    membershipChanged[threadIdx.x] = 0;

    //  BEWARE: We can overrun our shared memory here if there are too many
    //  clusters or too many coordinates!
    for (int i = threadIdx.x; i < numClusters; i += blockDim.x) {
        for (int j = 0; j < numCoords; j++) {
            clusters[numCoords * i + j] = deviceClusters[numCoords * i + j];
        }
    }
    __syncthreads();

    int objectId = blockDim.x * blockIdx.x + threadIdx.x;

    if (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;

        /* find the cluster id that has min distance to object */
        index    = 0;
        min_dist = euclid_dist_2(numCoords, numObjs, objects, objectId, clusters);

        for (i=1; i<numClusters; i++) {
            dist = euclid_dist_2(numCoords, numObjs, objects, objectId, clusters + numCoords * i);
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        /* assign the membership to object objectId */
        membership[objectId] = index;

        __syncthreads();    //  For membershipChanged[]

        //  blockDim.x *must* be a power of two!
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] +=
                    membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            intermediates[blockIdx.x] = membershipChanged[0];
        }
    }
}

__global__ static
void compute_delta(int *deviceIntermediates,
                   int numIntermediates,    //  The actual number of intermediates
                   int numIntermediates2)   //  The next power of two
{
    //  The number of elements in this array should be equal to
    //  numIntermediates2, the number of threads launched. It *must* be a power
    //  of two!
    extern __shared__ unsigned int intermediates[];

    //  Copy global intermediate values into shared memory.
    intermediates[threadIdx.x] =
        (threadIdx.x < numIntermediates) ? deviceIntermediates[threadIdx.x] : 0;

    __syncthreads();

    //  numIntermediates2 *must* be a power of two!
    for (unsigned int s = numIntermediates2 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            intermediates[threadIdx.x] += intermediates[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        deviceIntermediates[0] = intermediates[0];
    }
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
/* return an array of cluster centers of size [numClusters][numCoords]       */
float** cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
                   int     numCoords,    /* no. features */
                   int     numObjs,      /* no. objects */
                   int     numClusters,  /* no. clusters */
                   float   threshold,    /* % objects change membership */
                   int    *membership,   /* out: [numObjs] */
                   int    *loop_iterations)
{
    int      i, j, index, loop=0;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    float    delta;          /* % of objects change their clusters */
    float  **dimObjects;
    float  **clusters;       /* out: [numClusters][numCoords] */
    float  **newClusters;    /* [numClusters][numCoords] */

    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
    int *deviceIntermediates;
//  int *deviceNewClusterSize;

    malloc2D(dimObjects, numCoords, numObjs, float)
    for (i = 0; i < numCoords; i++) {
        for (j = 0; j < numObjs; j++) {
            dimObjects[i][j] = objects[j][i];
        }
    }

    /* allocate a 2D space for returning variable clusters[] (coordinates
       of cluster centers) */
    malloc2D(clusters, numClusters, numCoords, float)

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++)
            clusters[i][j] = objects[i][j];

    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    malloc2D(newClusters, numClusters, numCoords, float)

    //  To support reduction, numThreadsPerClusterBlock *must* be a power of
    //  two, and it *must* be no larger than the number of bits that will
    //  fit into an unsigned char, the type used to keep track of membership
    //  changes in the kernel.
    const unsigned int numThreadsPerClusterBlock = 128;
    const unsigned int numClusterBlocks =
        (numObjs + numThreadsPerClusterBlock - 1) / numThreadsPerClusterBlock;
    const unsigned int clusterBlockSharedDataSize =
        numThreadsPerClusterBlock * sizeof(unsigned char) +
        numClusters * numCoords * sizeof(float);

    const unsigned int numReductionThreads =
        nextPowerOfTwo(numClusterBlocks);
    const unsigned int reductionBlockSharedDataSize =
        numReductionThreads * sizeof(unsigned int);

    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
    checkCuda(hipMalloc(&deviceIntermediates, numReductionThreads*sizeof(unsigned int)));
//  checkCuda(hipMalloc(&deviceNewClusterSize, numClusters*sizeof(int)));

    checkCuda(hipMemcpy(deviceObjects, dimObjects[0],
              numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
//  checkCuda(hipMemcpy(deviceNewClusterSize, newClusterSize,
//            numClusters*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, clusters[0],
                  numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

//      msg("--- CUDA time! ---\n")
        find_nearest_cluster
            <<< numClusterBlocks, numThreadsPerClusterBlock, clusterBlockSharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceIntermediates);

        hipDeviceSynchronize(); checkLastCudaError();

        compute_delta <<< 1, numReductionThreads, reductionBlockSharedDataSize >>>
            (deviceIntermediates, numClusterBlocks, numReductionThreads);

        hipDeviceSynchronize(); checkLastCudaError();

        int d;
        checkCuda(hipMemcpy(&d, deviceIntermediates,
                  sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;
//      msg("%2d: delta = %d\n", loop, d)

//      checkCuda(hipMemcpy(clusters[0], deviceClusters,
//            numClusters*numCoords*sizeof(float), hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(membership, deviceMembership,
                  numObjs*sizeof(int), hipMemcpyDeviceToHost));
//      checkCuda(hipMemcpy(newClusterSize, deviceNewClusterSize,
//                numClusters*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
//          msg("membership[%d] = %d\n", i, membership[i])

            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i][j];
        }

        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i][j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }
            
        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    *loop_iterations = loop + 1;

    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
    checkCuda(hipFree(deviceIntermediates));
//  checkCuda(hipFree(deviceNewClusterSize));

    free(dimObjects[0]);
    free(dimObjects);
    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

