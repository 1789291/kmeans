#include "hip/hip_runtime.h"
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */
/*   File:         cuda_kmeans.cu  (CUDA version)                            */
/*   Description:  Implementation of simple k-means clustering algorithm     */
/*                 This program takes an array of N data objects, each with  */
/*                 M coordinates and performs a k-means clustering given a   */
/*                 user-provided value of the number of clusters (K). The    */
/*                 clustering results are saved in 2 arrays:                 */
/*                 1. a returned array of size [K][N] indicating the center  */
/*                    coordinates of K clusters                              */
/*                 2. membership[N] stores the cluster center ids, each      */
/*                    corresponding to the cluster a data object is assigned */
/*                                                                           */
/*   Author:  Wei-keng Liao                                                  */
/*            ECE Department, Northwestern University                        */
/*            email: wkliao@ece.northwestern.edu                             */
/*   Copyright, 2005, Wei-keng Liao                                          */
/*                                                                           */
/* * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * * */

#include <stdio.h>
#include <stdlib.h>

#include "kmeans.h"

static inline int nextPowerOfTwo(int n) {
    n--;

    n = n >>  1 | n;
    n = n >>  2 | n;
    n = n >>  4 | n;
    n = n >>  8 | n;
    n = n >> 16 | n;
//  n = n >> 32 | n;    //  For 64-bit ints

    return ++n;
}

/*----< euclid_dist_2() >----------------------------------------------------*/
/* square of Euclid distance between two multi-dimensional points            */
__host__ __device__ inline static
float euclid_dist_2(int    numdims,  /* no. dimensions */
                    float *coord1,   /* [numdims] */
                    float *coord2)   /* [numdims] */
{
    int i;
    float ans=0.0;

    for (i=0; i<numdims; i++)
        ans += (coord1[i]-coord2[i]) * (coord1[i]-coord2[i]);

    return(ans);
}

/*----< find_nearest_cluster() >---------------------------------------------*/
__global__ static
void find_nearest_cluster(int numCoords,
                          int numObjs,
                          int numClusters,
                          float *objects,   //  [numObjs][numCoords]
                          float *clusters,  //  [numObjs][numCoords]
                          int *membership,  //  [numObjs]
                          int *delta)
{
    //  The type chosen here must be large enough to support reductions!
    extern __shared__ unsigned short membershipChanged[];

    int objectId = blockDim.x * blockIdx.x + threadIdx.x;
    float *object = objects + numCoords * objectId;

    membershipChanged[threadIdx.x] = 0;

    if (objectId < numObjs) {
        int   index, i;
        float dist, min_dist;

        /* find the cluster id that has min distance to object */
        index    = 0;
        min_dist = euclid_dist_2(numCoords, object, clusters);

        for (i=1; i<numClusters; i++) {
            dist = euclid_dist_2(numCoords, object, clusters + numCoords * i);
            /* no need square root */
            if (dist < min_dist) { /* find the min and its array index */
                min_dist = dist;
                index    = i;
            }
        }

        if (membership[objectId] != index) {
            membershipChanged[threadIdx.x] = 1;
        }

        /* assign the membership to object objectId */
        membership[objectId] = index;

        __syncthreads();    //  For membershipChanged[]

        //  blockDim.x *must* be a power of two!
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                membershipChanged[threadIdx.x] += membershipChanged[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            *delta = membershipChanged[0];
        }
    }
}

/*----< cuda_kmeans() >-------------------------------------------------------*/
/* return an array of cluster centers of size [numClusters][numCoords]       */
float** cuda_kmeans(float **objects,      /* in: [numObjs][numCoords] */
                   int     numCoords,    /* no. features */
                   int     numObjs,      /* no. objects */
                   int     numClusters,  /* no. clusters */
                   float   threshold,    /* % objects change membership */
                   int    *membership)   /* out: [numObjs] */
{
    int      i, j, index, loop=1;
    int     *newClusterSize; /* [numClusters]: no. objects assigned in each
                                new cluster */
    float    delta;          /* % of objects change their clusters */
    float  **clusters;       /* out: [numClusters][numCoords] */
    float  **newClusters;    /* [numClusters][numCoords] */

    int *deviceDelta;
    float *deviceObjects;
    float *deviceClusters;
    int *deviceMembership;
//  int *deviceNewClusterSize;

    /* allocate a 2D space for returning variable clusters[] (coordinates
       of cluster centers) */
    clusters    = (float**) malloc(numClusters *             sizeof(float*));
    assert(clusters != NULL);
    clusters[0] = (float*)  malloc(numClusters * numCoords * sizeof(float));
    assert(clusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        clusters[i] = clusters[i-1] + numCoords;

    /* pick first numClusters elements of objects[] as initial cluster centers*/
    for (i=0; i<numClusters; i++)
        for (j=0; j<numCoords; j++)
            clusters[i][j] = objects[i][j];

    /* initialize membership[] */
    for (i=0; i<numObjs; i++) membership[i] = -1;

    /* need to initialize newClusterSize and newClusters[0] to all 0 */
    newClusterSize = (int*) calloc(numClusters, sizeof(int));
    assert(newClusterSize != NULL);

    newClusters    = (float**) malloc(numClusters *            sizeof(float*));
    assert(newClusters != NULL);
    newClusters[0] = (float*)  calloc(numClusters * numCoords, sizeof(float));
    assert(newClusters[0] != NULL);
    for (i=1; i<numClusters; i++)
        newClusters[i] = newClusters[i-1] + numCoords;

    checkCuda(hipMalloc(&deviceDelta, sizeof(int)));
    checkCuda(hipMalloc(&deviceObjects, numObjs*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceClusters, numClusters*numCoords*sizeof(float)));
    checkCuda(hipMalloc(&deviceMembership, numObjs*sizeof(int)));
//  checkCuda(hipMalloc(&deviceNewClusterSize, numClusters*sizeof(int)));

    checkCuda(hipMemcpy(deviceObjects, objects[0],
              numObjs*numCoords*sizeof(float), hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(deviceMembership, membership,
              numObjs*sizeof(int), hipMemcpyHostToDevice));
//  checkCuda(hipMemcpy(deviceNewClusterSize, newClusterSize,
//            numClusters*sizeof(int), hipMemcpyHostToDevice));

    do {
        checkCuda(hipMemcpy(deviceClusters, clusters[0],
                  numClusters*numCoords*sizeof(float), hipMemcpyHostToDevice));

        const unsigned int numThreadsPerBlock = 128;
//      const unsigned int numThreadsPerBlock = nextPowerOfTwo(numObjs);
        const unsigned int numBlocks = (numObjs + numThreadsPerBlock - 1) / numThreadsPerBlock;
//      const unsigned int numBlocks = 1;
        const unsigned int sharedDataSize = numThreadsPerBlock * sizeof(unsigned short);

        msg("--- CUDA time! ---\n")
        find_nearest_cluster <<< numBlocks, numThreadsPerBlock, sharedDataSize >>>
            (numCoords, numObjs, numClusters,
             deviceObjects, deviceClusters, deviceMembership, deviceDelta);

        hipDeviceSynchronize();
        checkLastCudaError();

        int d;
        checkCuda(hipMemcpy(&d, deviceDelta,
                  sizeof(int), hipMemcpyDeviceToHost));
        delta = (float)d;
        msg("Delta = %d\n", d)

        checkCuda(hipMemcpy(clusters[0], deviceClusters,
              numClusters*numCoords*sizeof(float), hipMemcpyDeviceToHost));
        checkCuda(hipMemcpy(membership, deviceMembership,
                  numObjs*sizeof(int), hipMemcpyDeviceToHost));
//      checkCuda(hipMemcpy(newClusterSize, deviceNewClusterSize,
//                numClusters*sizeof(int), hipMemcpyDeviceToHost));

        for (i=0; i<numObjs; i++) {
            /* find the array index of nestest cluster center */
            index = membership[i];
//          msg("membership[%d] = %d\n", i, membership[i])

            /* update new cluster centers : sum of objects located within */
            newClusterSize[index]++;
            for (j=0; j<numCoords; j++)
                newClusters[index][j] += objects[i][j];
        }

        /* average the sum and replace old cluster centers with newClusters */
        for (i=0; i<numClusters; i++) {
            for (j=0; j<numCoords; j++) {
                if (newClusterSize[i] > 0)
                    clusters[i][j] = newClusters[i][j] / newClusterSize[i];
                newClusters[i][j] = 0.0;   /* set back to 0 */
            }
            newClusterSize[i] = 0;   /* set back to 0 */
        }
            
        delta /= numObjs;
    } while (delta > threshold && loop++ < 500);

    msg("Loop iterations: %d\n", loop)

    checkCuda(hipFree(deviceDelta));
    checkCuda(hipFree(deviceObjects));
    checkCuda(hipFree(deviceClusters));
    checkCuda(hipFree(deviceMembership));
//  checkCuda(hipFree(deviceNewClusterSize));

    free(newClusters[0]);
    free(newClusters);
    free(newClusterSize);

    return clusters;
}

